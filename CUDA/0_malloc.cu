
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
	int in[3] = {1,2,3};
	int out[3]  = {0,};

	int* Gmem;
	
	hipMalloc((void**)&Gmem,3*sizeof(int));
	

	//int -> Gmem
	hipMemcpy(Gmem, in, 3 * sizeof(int), hipMemcpyHostToDevice );

	//Gmem -> out
	hipMemcpy(out, Gmem, 3 * sizeof(int) ,hipMemcpyDeviceToHost );

	for(int i=0;i<3;i++)
 		printf("%d\n",out[i]);

	hipFree(Gmem);

	return 0;
}
