
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CHECK 0
#define RESULT 0

#define DATA_TYPE double
#define BLOCK_NUM 1
#define THREAD_NUM 1
#define MATRIX_NUM 10
#define MATRIX_SIZE 10
#define ITER int
void stopwatch(int);

void cublas_error();
void pp(int p)
{printf("------------ %d-------------\n",p);}

void mat_out(DATA_TYPE*);

int main()
{
	printf("BLOCK_NUM :%d\nTHREAD_NUM : %d\nMATRIX_NUM : %d\nMATRIX_SIZE : (%d)X(%d)\n",BLOCK_NUM,THREAD_NUM,MATRIX_NUM,MATRIX_SIZE,MATRIX_SIZE);

	//host matrix array
	DATA_TYPE *h_a,*h_b,*h_c;
	//device matrix array
	DATA_TYPE *d_a,*d_b,*d_c;
	
	//blas parameters
	DATA_TYPE alpha=1,beta=0;
	int m,n,k,lda,ldb,ldc;
	hipblasOperation_t transa,transb;
	
	long long stridea,strideb,stridec;

	//matrix sizes
	long long s2;
	long long s3;
//	cublasHandle_t handle;
	
	int offset[MATRIX_NUM];

	hipblasHandle_t handle;

	hipblasHandle_t handle_s[MATRIX_NUM];
	hipStream_t stream[MATRIX_NUM];

	//디버그
	hipblasStatus_t cublas_stat;
	hipError_t cuda_stat;

/************************Initialization******************************************/	

	m=MATRIX_SIZE,n=MATRIX_SIZE,k=MATRIX_SIZE,lda=MATRIX_SIZE,ldb=MATRIX_SIZE,ldc=MATRIX_SIZE;
	s2 = MATRIX_SIZE * MATRIX_SIZE;
	s3 = MATRIX_SIZE * MATRIX_SIZE * MATRIX_SIZE;	

	transa = HIPBLAS_OP_N;
	transb = HIPBLAS_OP_N;	

	stridea = s2;
	strideb = s2;
	stridec = s2;
	
	cuda_stat = hipHostMalloc((void**)&h_a,sizeof(DATA_TYPE) * MATRIX_NUM*s3, hipHostMallocDefault);

#if CHECK
	printf("cudaMallocHost : %d\n",cuda_stat);
#endif	
	cuda_stat=	hipHostMalloc((void**)&h_b,sizeof(DATA_TYPE) * MATRIX_NUM*s3, hipHostMallocDefault);
#if CHECK
	printf("cudaMallocHost : %d\n",cuda_stat);
#endif
	cuda_stat=	hipHostMalloc((void**)&h_c,sizeof(DATA_TYPE) * MATRIX_NUM*s3, hipHostMallocDefault);

	hipMalloc((void**)&d_a,sizeof(DATA_TYPE)*MATRIX_NUM*s3);
	hipMalloc((void**)&d_b,sizeof(DATA_TYPE)*MATRIX_NUM*s3);
	hipMalloc((void**)&d_c,sizeof(DATA_TYPE)*MATRIX_NUM*s3);


	srand(time(NULL));

	hipDeviceSynchronize();
	

	for(long long j=0;j<s3;j++)h_a[j]=rand()/(DATA_TYPE)RAND_MAX;
    for(long long j=0;j<s3;j++)h_b[j]=rand()/(DATA_TYPE)RAND_MAX;
	for(long long j=0;j<s3;j++)h_c[j]=0;

	hipblasCreate(&handle);


	for(int i=0;i<MATRIX_NUM;i++)
		hipblasCreate(&(handle_s[i]));
	for(int i=0;i<MATRIX_NUM;i++)
		hipStreamCreate(&(stream[i]));
	for(int i=0;i<MATRIX_NUM;i++)
		hipblasSetStream(handle_s[i],stream[i]);



/****************** GEMM  한번 ********************/
/*
	printf("a GEMM : \n");
	stopwatch(0);

	cudaMemcpy(d_a,h_a,sizeof(DATA_TYPE)*s2,cudaMemcpyHostToDevice);
	cudaMemcpy(d_b,h_b,sizeof(DATA_TYPE)*s2,cudaMemcpyHostToDevice);

	cublasDgemm(handle,transa,transb,m,n,k,&alpha,d_a,lda,d_b,ldb,&beta,d_c,ldc);	
	
	cudaMemcpy(h_c,d_c,sizeof(DATA_TYPE)*s2,cudaMemcpyDeviceToHost);

	stopwatch(1);


*/
/******************그냥 GEMM  ********************/
	
	printf("GEMMs : \n");
	stopwatch(0);

	hipMemcpy(d_a,h_a,sizeof(DATA_TYPE)*s3,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeof(DATA_TYPE)*s3,hipMemcpyHostToDevice);

	for(int i=0;i<MATRIX_NUM;i++)
		offset[i] = i * s2;

	for(int i=0;i<MATRIX_NUM;i++)
{
	cublas_stat =	hipblasDgemm(handle,transa,transb,m,n,k,&alpha,&d_a[offset[i]],lda,&d_b[offset[i]],ldb,&beta,&d_c[offset[i]],ldc);	

#if CHECK
	printf("DGEMM[%d] : %d\n",offset[i],cublas_stat);
#endif

}	
		hipMemcpy(h_c,d_c,sizeof(DATA_TYPE)*s3,hipMemcpyDeviceToHost);

	stopwatch(1);
#if RESULT
	mat_out(h_c);
#endif

	for(int i=0;i<MATRIX_NUM;i++)
		for(int j=0;j<MATRIX_SIZE;j++)
			for(int k=0;k<MATRIX_SIZE;k++)
				h_c[i*s2 + j*MATRIX_SIZE + k] = 0 ;



/******************BATCHED STRIDE GEMM  ********************/

	printf("BATCHED GEMM : \n");
	stopwatch(0);

	hipMemcpy(d_a,h_a,sizeof(DATA_TYPE)*s3,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeof(DATA_TYPE)*s3,hipMemcpyHostToDevice);

cublas_stat=	hipblasDgemmStridedBatched(handle,transa,transb,m,n,k,&alpha,d_a,lda,stridea,d_b,ldb,strideb,&beta,d_c,ldc,stridec,MATRIX_NUM);	
#if CHECK	
	printf("Dgemm Strided Bached : %d\n",cublas_stat);
#endif
	hipMemcpy(h_c,d_c,sizeof(DATA_TYPE)*s3,hipMemcpyDeviceToHost);

	stopwatch(1);
#if RESULT
	mat_out(h_c);
#endif

	for(int i=0;i<MATRIX_NUM;i++)
		for(int j=0;j<MATRIX_SIZE;j++)
			for(int k=0;k<MATRIX_SIZE;k++)
				h_c[i*s2 + j*MATRIX_SIZE + k] = 0 ;





/******************BATCHED GEMM  ********************/
/*
	printf("BATCH : \n");
	stopwatch(0);

	for(int i=0;i<MATRIX_NUM;i++)
		cudaMemcpy(d_a[i],h_a[i],sizeof(DATA_TYPE)*s2,cudaMemcpyHostToDevice);
	for(int i=0;i<MATRIX_NUM;i++)
		cudaMemcpy(d_b[i],h_b[i],sizeof(DATA_TYPE)*s2,cudaMemcpyHostToDevice);

	cublas_stat = cublasDgemmBatched(handle,transa,transb,m,n,k,&alpha,(const DATA_TYPE**)d_a,lda,(const DATA_TYPE**)d_b,ldb,&beta,d_c,ldc,MATRIX_NUM);	
#if CHECK
	printf("cublasDgemmBatched : %d\n",cublas_stat);
#endif

	
	for(int i=0;i<MATRIX_NUM;i++)
		cudaMemcpy(h_c[i],d_c[i],sizeof(DATA_TYPE)*s2,cudaMemcpyDeviceToHost);

	stopwatch(1);
#if CHECK
	mat_out(h_c);
#endif

*/
//	cublasDestroy(handle);





/******************STREAMED GEMM  ********************/
	hipDeviceSynchronize();

	
	printf("STREAM : \n");
	stopwatch(0);


	for(int i=0;i<MATRIX_NUM;i++)
{	 cuda_stat = hipMemcpyAsync(&d_a[offset[i]],&h_a[offset[i]],sizeof(DATA_TYPE)*s2,hipMemcpyHostToDevice,stream[i]);

#if CHECK
	printf("cudaMemcpyAsync[%d] : %d\n",i,cuda_stat);
#endif
}	
	for(int i=0;i<MATRIX_NUM;i++)
		hipMemcpyAsync(&d_b[offset[i]],&h_b[offset[i]],sizeof(DATA_TYPE)*s2,hipMemcpyHostToDevice,stream[i]);

	for(int i=0;i<MATRIX_NUM;i++)
{		cublas_stat =	hipblasDgemm(handle_s[i],transa,transb,m,n,k,&alpha,&d_a[offset[i]],lda,&d_b[offset[i]],ldb,&beta,&d_c[offset[i]],ldc);	
	
#if CHECK
	printf("cublasDgemm : %d\n",cublas_stat);

#endif
}

	
	for(int i=0;i<MATRIX_NUM;i++)
		hipMemcpyAsync(&h_c[offset[i]],&d_c[offset[i]],sizeof(DATA_TYPE)*s2,hipMemcpyDeviceToHost,stream[i]);

	for(int i=0;i<MATRIX_NUM;i++)
		hipStreamSynchronize(stream[i]);
	stopwatch(1);


#if RESULT
	mat_out(h_c);
#endif


/***********DeAllocation**********************/
	
	hipFree(h_a);
	hipFree(h_b);
	hipFree(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipblasDestroy(handle);

	
	for(int i=0;i<MATRIX_NUM;i++)
		hipblasDestroy(handle_s[i]);
	for(int i=0;i<MATRIX_NUM;i++)
		hipStreamDestroy(stream[i]);		

	return 0;
}

void stopwatch(int flag)
{
	enum clock_unit{nano = 0, micro , milli, sec} unit;
	
	const long long NANOS = 1000000000LL;
	static struct timespec startTS,endTS;
	static long long diff = 0;

	/*
		여기서 단위 조정
		nano, micro, milli, sec
	*/
	unit = micro;

	//start
	if(flag == 0)
	{
		diff = 0;
		if(-1 == clock_gettime(CLOCK_MONOTONIC,&startTS))
			printf("Failed to call clock_gettime\n");
	}
	//end
	else if(flag == 1)
	{		
		if(-1 == clock_gettime(CLOCK_MONOTONIC,&endTS))
			printf("Failed to call clock_gettime\n");
		diff = NANOS * (endTS.tv_sec - startTS.tv_sec) + (endTS.tv_nsec - startTS.tv_nsec);

		switch(unit)		
		{
			case nano :
				printf("elapsed time : % lld nano sec\n",diff);
			break;
			case micro :
				printf("elapsed time : % lld micro sec\n",diff/1000);
			break;
			case sec :
				printf("elapsed time : % lld sec\n",diff/1000000000);
			break;
			default :
				printf("elapsed time : % lld milli sec\n",diff/100000);
			break;	

		}
	}
	else
	{
		printf("wrong flag | 0 : start, 1 : end\n");
	}

}


void mat_out(DATA_TYPE*a)
{
	for(int i=0;i<MATRIX_NUM;i++)
	{
		printf("--- %d ---\n",i);
		
		for(int j=0;j<MATRIX_SIZE;j++)
		{
			for(int k=0;k<MATRIX_SIZE;k++)
			{
				printf("%.3lf ",a[i*MATRIX_SIZE*MATRIX_SIZE + j*MATRIX_SIZE + k]);
			}
			printf("\n");
		
		}





	}
	

}
